#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "common.h"
#include "cuda_common.cuh"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>


__global__ void sum_array_gpu(int* a, int* b, int* c, int size){
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    if (gid < size) {
        c[gid] = a[gid] + b[gid];
    }

}

void sum_array_cpu(int* a, int* b, int* c, int size) {
    for (int i = 0; i < size; i++) {
        c[i] = a[i] + b[i];
    }
}

int main(){
    
    int size = 2000;
    int block_size = 128;

    int NO_BYTES = size * sizeof(int);

    //host pointers
    int* h_a, * h_b, * gpu_results, *h_c;

    h_a = (int*)malloc(NO_BYTES);
    h_b = (int*)malloc(NO_BYTES);
    gpu_results = (int*)malloc(NO_BYTES);
    h_c = (int*)malloc(NO_BYTES);



    time_t t;
    srand((unsigned)time(&t));

    for (int i = 0; i < size; i++) {
        h_a[i] = (int)(rand() & 0xff);
        h_b[i] = (int)(rand() & 0xff);
    }

    //sum in cpu
    clock_t cpu_start, cpu_end;

    cpu_start = clock();
    sum_array_cpu(h_a, h_b, h_c, size);
    cpu_end = clock();

    //device pointer
    int* d_a, * d_b, * d_c;
    hipError_t error;

    gpuErrchk(hipMalloc((int**)&d_a, NO_BYTES));
    gpuErrchk(hipMalloc((int**)&d_b, NO_BYTES));
    gpuErrchk(hipMalloc((int**)&d_c, NO_BYTES));

    //monitor transfer time from host to device
    clock_t htod_start, htod_end;
    htod_start = clock();
    
    //memory transfer from host to device
    hipMemcpy(d_a, h_a, NO_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, NO_BYTES, hipMemcpyHostToDevice);
    
    htod_end = clock();

    //kernel launch parameter
    dim3 block(block_size);
    dim3 grid((size / block.x) + 1);
     


    //sum in GPU
    clock_t gpu_start, gpu_end;
    gpu_start = clock();
    sum_array_gpu << < grid, block >> > (d_a, d_b, d_c, size);
    hipDeviceSynchronize();
    gpu_end = clock();

    //transfer results back to the host. Time the tranfer latency
    clock_t dtoh_start, dtoh_end;
    dtoh_start = clock();
    hipMemcpy(gpu_results, d_c, NO_BYTES, hipMemcpyDeviceToHost);
    dtoh_end = clock();

    //array comparisons between cpu and gpu calculation
    compare_arrays(gpu_results, h_c, size);
    
    printf("Sum array CPU execution time : %4.6f \n",(double)((double)(cpu_end - cpu_start)/CLOCKS_PER_SEC));
    printf("Sum array GPU execution time : %4.6f \n",(double)((double)(gpu_end - gpu_start)/CLOCKS_PER_SEC));
    printf("htod mem transfer time : %4.6f \n",(double)((double)(htod_end - htod_start)/CLOCKS_PER_SEC));
    printf("dtoh mem transfer time : %4.6f \n",(double)((double)(dtoh_end - dtoh_start)/CLOCKS_PER_SEC));
    printf("Sum array GPU total execution time : %4.6f \n",(double)((double)(dtoh_end - htod_start)/CLOCKS_PER_SEC));

    hipFree(d_c);
    hipFree(d_b);
    hipFree(d_a);

    free(h_a);
    free(h_b);
    free(gpu_results);

    hipDeviceReset();
    return 0;

}
